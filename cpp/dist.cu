#include <hip/hip_runtime.h>

#include <vector>
#include <iostream>

#define check(ans) { _check((ans), __FILE__, __LINE__); }
inline void _check(hipError_t code, char* file, int line)
{
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

int div_up(int a, int b) {
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

__global__ void calc_distances_kernel(double* distances, double* atoms, int num_dimensions, int M);

__global__ void calc_distances_kernel(double* distances, double* atoms, int num_dimensions, int M)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= M) return;

    for (int j = idx + 1; j < M; ++j) {
        double sum = 0;
        for (int k = 0; k < num_dimensions; ++k) {
            sum+= (atoms[idx * num_dimensions + k] - atoms[j * num_dimensions + k]) * (atoms[idx * num_dimensions + k] - atoms[j * num_dimensions + k]);
        }
        distances[idx * M + j] = sqrt(sum);
        distances[j * M + idx] = sqrt(sum);
    }

}

void calculate_distance_matrix_cuda(double *input_ij_1d, double *output_ij_1d, int num_dimensions, int M) {
    double* distances_d;
    double* all_points_d;

    check(hipMalloc(&distances_d, M * M * sizeof(double)));
    check(hipMalloc(&all_points_d, M * num_dimensions * sizeof(double)));
    check(hipMemcpy(all_points_d, input_ij_1d, M * num_dimensions * sizeof(double), hipMemcpyHostToDevice));

    const int threads_per_block(512);
    dim3 n_blocks(div_up(M, threads_per_block));

    calc_distances_kernel <<< n_blocks, threads_per_block >>> (distances_d, all_points_d, num_dimensions, M);

    check(hipPeekAtLastError());
    check(hipDeviceSynchronize());


    check(hipMemcpy(output_ij_1d, distances_d, M * M * sizeof(double), hipMemcpyDeviceToHost));


    check(hipFree(all_points_d));
    check(hipFree(distances_d));
}
