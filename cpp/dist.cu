#include <hip/hip_runtime.h>

#include <vector>
#include <iostream>
const int N(10);

#define check(ans) { _check((ans), __FILE__, __LINE__); }
inline void _check(hipError_t code, char* file, int line)
{
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

int div_up(int a, int b) {
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

__global__ void calc_distances_kernel(double* distances, double** atoms, int num_dimensions);

__global__ void calc_distances_kernel(double* distances, double* atoms, int num_dimensions)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N) return;

    for (int j = idx + 1; j < N; ++j) {
        double sum = 0;
        for (int k = 0; k < num_dimensions; ++k) {
            sum+= (atoms[idx * num_dimensions + k] - atoms[j * num_dimensions + k]) * (atoms[idx * num_dimensions + k] - atoms[j * num_dimensions + k]);
        }
        distances[idx * N + j] = sqrt(sum);
        distances[j * N + idx] = sqrt(sum);
    }
}

void calculate_distance_matrix_cuda(double *input_ij_1d, double *output_ij_1d, int num_dimensions) {
    double* distances_d;
    double* all_points_d;

    check(hipMalloc(&distances_d, N * N * sizeof(double)));
    check(hipMalloc(&all_points_d, N * num_dimensions * sizeof(double)));
    check(hipMemcpy(all_points_d, input_ij_1d, N * num_dimensions * sizeof(double), hipMemcpyHostToDevice));

    const int threads_per_block(512);
    dim3 n_blocks(div_up(N, threads_per_block));

    calc_distances_kernel <<< n_blocks, threads_per_block >>> (distances_d, all_points_d, num_dimensions);

    check(hipPeekAtLastError());
    check(hipDeviceSynchronize());


    check(hipMemcpy(output_ij_1d, distances_d, N * N * sizeof(double), hipMemcpyDeviceToHost));


    check(hipFree(all_points_d));
    check(hipFree(distances_d));
}
